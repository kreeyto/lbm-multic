/* 
 * ------------------------------------------------------------
 * Author: Breno Vargas Gemelgo
 * CUDA Implementation: Lattice Boltzmann Method
 * Module: Moments and Collision Step for Bubble Simulation
 * 
 * To compile and integrate with MATLAB, use the following:
 * nvcc -ptx gpuMomCollision.cu -o gpuMomCollision.ptx
 * ------------------------------------------------------------
 */

#include <hip/hip_runtime.h>
#include <math.h>

__global__ void momCollision(
        double *rho, double *ux, double *uy, double *uz,  
        double *ffx, double *ffy, double *ffz, double *f,         
        int nx, int ny, int nz, 
        double cssq, double *cix, double *ciy, double *ciz, double *w,
        double *pxx, double *pyy, double *pzz, double *pxy, double *pxz, double *pyz,
        int fpoints,
        double omega, double sharp_c, double *w_g, double *phi,
        double *normx, double *normy, double *normz, double *g,
        int gpoints
    ) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i + nx * (j + ny * k);
    #define F_IDX(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    // moments
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        ux[idx] = (
            (f[F_IDX(i,j,k,1)] + f[F_IDX(i,j,k,15)] + f[F_IDX(i,j,k,9)] + f[F_IDX(i,j,k,7)] + f[F_IDX(i,j,k,13)]) -
            (f[F_IDX(i,j,k,2)] + f[F_IDX(i,j,k,10)] + f[F_IDX(i,j,k,16)] + f[F_IDX(i,j,k,14)] + f[F_IDX(i,j,k,7)])
        ) / rho[idx] +
        ffx[idx] * 0.5 / rho[idx];
        uy[idx] = (
            (f[F_IDX(i,j,k,3)] + f[F_IDX(i,j,k,7)] + f[F_IDX(i,j,k,14)] + f[F_IDX(i,j,k,17)] + f[F_IDX(i,j,k,11)]) -
            (f[F_IDX(i,j,k,4)] + f[F_IDX(i,j,k,13)] + f[F_IDX(i,j,k,8)] + f[F_IDX(i,j,k,12)] + f[F_IDX(i,j,k,18)])
        ) / rho[idx] +
        ffy[idx] * 0.5 / rho[idx];
        uz[idx] = (
            (f[F_IDX(i,j,k,6)] + f[F_IDX(i,j,k,15)] + f[F_IDX(i,j,k,10)] + f[F_IDX(i,j,k,17)] + f[F_IDX(i,j,k,12)]) -
            (f[F_IDX(i,j,k,5)] + f[F_IDX(i,j,k,9)] + f[F_IDX(i,j,k,16)] + f[F_IDX(i,j,k,11)] + f[F_IDX(i,j,k,18)])
        ) / rho[idx] +
        ffz[idx] * 0.5 / rho[idx];

        double fneq[19];
        double uu = 0.5 * (pow(ux[idx],2) + pow(uy[idx],2) + pow(uz[idx],2)) / cssq;

        for (int n = 0; n < fpoints; n++) { 
            rho[idx] += f[idx + n * nx * ny * nz]; 
        }

        for (int l = 0; l < fpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double HeF = (w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu)))
                     * ((cix[l] - ux[idx]) * ffx[idx] + 
                        (ciy[l] - uy[idx]) * ffy[idx] + 
                        (ciz[l] - uz[idx]) * ffz[idx] 
                       ) / (rho[idx] * cssq);
            double feq = w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu)) - 0.5 * HeF;
            fneq[l] = f[F_IDX(i,j,k,l)] - feq;
        }

        pxx[idx] = fneq[2] + fneq[3] + fneq[8] + fneq[9] + fneq[10] + fneq[11] + fneq[14] + fneq[15] + fneq[16] + fneq[17];
        pyy[idx] = fneq[4] + fneq[5] + fneq[8] + fneq[9] + fneq[12] + fneq[13] + fneq[14] + fneq[15] + fneq[18] + fneq[19];
        pzz[idx] = fneq[6] + fneq[7] + fneq[10] + fneq[11] + fneq[12] + fneq[13] + fneq[16] + fneq[17] + fneq[18] + fneq[19];
        pxy[idx] = fneq[8] + fneq[9] - fneq[14] - fneq[15];
        pxz[idx] = fneq[10] + fneq[11] - fneq[16] - fneq[17];
        pyz[idx] = fneq[12] + fneq[13] - fneq[18] - fneq[19];

    }

    // collision
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        double uu = 0.5 * (pow(ux[idx],2) + pow(uy[idx],2) + pow(uz[idx],2)) / cssq;

        for (int l = 0; l < fpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double feq = w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu));
            double HeF = 0.5 * (w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu)))
                     * ((cix[l] - ux[idx]) * ffx[idx] + 
                        (ciy[l] - uy[idx]) * ffy[idx] + 
                        (ciz[l] - uz[idx]) * ffz[idx] 
                       ) / (rho[idx] * cssq);
            double fneq = (cix[l] * cix[l] - cssq) * pxx[idx] + 
                          (ciy[l] * ciy[l] - cssq) * pyy[idx] + 
                          (ciz[l] * ciz[l] - cssq) * pzz[idx] + 
                          2 * cix[l] * ciy[l] * pxy[idx] + 
                          2 * cix[l] * ciz[l] * pxz[idx] + 
                          2 * ciy[l] * ciz[l] * pyz[idx];
            f[F_IDX(i+static_cast<int>(cix[l]),
                    j+static_cast<int>(ciy[l]),
                    k+static_cast<int>(ciz[l]),
                    l)] = feq + (1-omega) * (w[l] / (2*pow(cssq,2))) * fneq + HeF;
        }

        for (int l = 0; l < gpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double feq = w_g[l] * phi[idx] * (1 + udotc);
            double Hi = sharp_c * phi[idx] * (1 - phi[idx]) * (cix[l] * normx[idx] + ciy[l] * normy[idx] + ciz[l] * normz[idx]); 
            g[F_IDX(i,j,k,l)] = feq + w_g[l] * Hi;
        }

    }
}

