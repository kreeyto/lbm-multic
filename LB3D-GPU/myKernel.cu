#include <hip/hip_runtime.h>
#include <math.h>

    /*  nvcc -ptx myKernel.cu -o myKernel.ptx  */

__global__ void momCollision(
        double *rho, double *ux, double *uy, double *uz,  
        double *ffx, double *ffy, double *ffz, double *f,         
        int nx, int ny, int nz, 
        double cssq, double *cix, double *ciy, double *ciz, double *w,
        double *pxx, double *pyy, double *pzz, double *pxy, double *pxz, double *pyz,
        int fpoints
    ) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // moments
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {
        int idx = i + nx * (j + ny * k);
        #define F_INDEX(i,j,k,q) ((i) + nx * ((j) + ny * ((k) + nz * (q))))

        ux[idx] = (
            (f[F_INDEX(i,j,k,1)] + f[F_INDEX(i,j,k,15)] + f[F_INDEX(i,j,k,9)] + f[F_INDEX(i,j,k,7)] + f[F_INDEX(i,j,k,13)]) -
            (f[F_INDEX(i,j,k,2)] + f[F_INDEX(i,j,k,10)] + f[F_INDEX(i,j,k,16)] + f[F_INDEX(i,j,k,14)] + f[F_INDEX(i,j,k,7)])
        ) / rho[idx] +
        ffx[idx] * 0.5 / rho[idx];
        uy[idx] = (
            (f[F_INDEX(i,j,k,3)] + f[F_INDEX(i,j,k,7)] + f[F_INDEX(i,j,k,14)] + f[F_INDEX(i,j,k,17)] + f[F_INDEX(i,j,k,11)]) -
            (f[F_INDEX(i,j,k,4)] + f[F_INDEX(i,j,k,13)] + f[F_INDEX(i,j,k,8)] + f[F_INDEX(i,j,k,12)] + f[F_INDEX(i,j,k,18)])
        ) / rho[idx] +
        ffy[idx] * 0.5 / rho[idx];
        uz[idx] = (
            (f[F_INDEX(i,j,k,6)] + f[F_INDEX(i,j,k,15)] + f[F_INDEX(i,j,k,10)] + f[F_INDEX(i,j,k,17)] + f[F_INDEX(i,j,k,12)]) -
            (f[F_INDEX(i,j,k,5)] + f[F_INDEX(i,j,k,9)] + f[F_INDEX(i,j,k,16)] + f[F_INDEX(i,j,k,11)] + f[F_INDEX(i,j,k,18)])
        ) / rho[idx] +
        ffz[idx] * 0.5 / rho[idx];

        double fneq[19];
        
        double uu = 0.5 * (pow(ux[idx],2) + pow(uy[idx],2) + pow(uz[idx],2)) / cssq;

        for (int n = 0; n < fpoints; n++) { 
            rho[idx] += f[idx + n * nx * ny * nz]; 
        }

        for (int l = 0; l < fpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double HeF = (w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu)))
                     * ((cix[l] - ux[idx]) * ffx[idx] + 
                        (ciy[l] - uy[idx]) * ffy[idx] + 
                        (ciz[l] - uz[idx]) * ffz[idx] 
                       ) / (rho[idx] * cssq);
            double feq = w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc,2) - uu)) - 0.5 * HeF;
            fneq[l] = f[F_INDEX(i,j,k,l)] - feq;
        }

        pxx[idx] = fneq[2] + fneq[3] + fneq[8] + fneq[9] + fneq[10] + fneq[11] + fneq[14] + fneq[15] + fneq[16] + fneq[17];
        pyy[idx] = fneq[4] + fneq[5] + fneq[8] + fneq[9] + fneq[12] + fneq[13] + fneq[14] + fneq[15] + fneq[18] + fneq[19];
        pzz[idx] = fneq[6] + fneq[7] + fneq[10] + fneq[11] + fneq[12] + fneq[13] + fneq[16] + fneq[17] + fneq[18] + fneq[19];
        pxy[idx] = fneq[8] + fneq[9] - fneq[14] - fneq[15];
        pxz[idx] = fneq[10] + fneq[11] - fneq[16] - fneq[17];
        pyz[idx] = fneq[12] + fneq[13] - fneq[18] - fneq[19];

    }

    // collision
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {
        // code for collision
        int idx = i + nx * (j + ny * k);
        #define F_INDEX(i,j,k,q) ((i) + nx * ((j) + ny * ((k) + nz * (q))))
    }
}

